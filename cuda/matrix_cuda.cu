#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matMulKernel(const double *mat1, const double *mat2, double *temp, size_t row, size_t mid, size_t col) {
    size_t r = blockIdx.y * blockDim.y + threadIdx.y;
    size_t c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < row && c < col) {
        double sum = 0.0;
        for (size_t k = 0; k < mid; k++) {
            sum += mat1[r * mid + k] * mat2[k * col + c];
        }
        temp[r * col + c] = sum;
    }
}

extern "C" void launchMatrixMultiply(const double* mat1, const double* mat2, double* result, 
                                    size_t row, size_t mid, size_t col) {
    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((col + blockDim.x - 1) / blockDim.x, 
                 (row + blockDim.y - 1) / blockDim.y);
    
    // Launch kernel
    matMulKernel<<<gridDim, blockDim>>>(mat1, mat2, result, row, mid, col);
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}